#include "hip/hip_runtime.h"
#include "converge.cuh"

#define SIZE 1024

/*__device__ Eigen::Matrix<ScalarType, Eigen::Dynamic, 1> p_spring;
__device__ Eigen::Matrix<ScalarType, Eigen::Dynamic, 1> p_attach;
__device__ Eigen::Matrix<ScalarType, Eigen::Dynamic, 1>* p_j; //May need to be put into the jacobiOnDevice function
__device__ Eigen::Matrix<ScalarType, Eigen::Dynamic, 1> q_n1;*/



__global__ void localStep(double * p_spring, double* p_attach, double* p_j, double* q_n1, double* b ) {
	//Constraint* cj;
	//ScalarType current_strecth; 
	//EigenVector3 current_vector;
	//int cSize = sizeof m_constraint / sizeof * m_constraint;
	int idx = threadIdx.x;
	int idy = threadIdx.y;
	//Parse in the constraint

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	//p_j is just a Matrix.

	//Getting cons

	/*for (int i = index; i < cSize; i += stride) {
		cj = &m_constraint[i];
		if (cj->constraintType == SPRING) {
			// Work out spring constraint.
			SpringConstraint* sc = (SpringConstraint*) cj;
			current_vector = q_n1->block_vector(sc->GetConstrainedVertexIndex1()) - q_n1->block_vector(sc->GetConstrainedVertexIndex2());
			current_strecth = current_vector.norm() - sc->GetRestLength();
			current_vector = (current_strecth / 2.0) * current_vector.normalized();

			p_j = p_spring;
			p_j->block_vector(0) = q_n1->block_vector(sc->GetConstrainedVertexIndex1()) - current_vector;
			p_j->block_vector(1) = q_n1->block_vector(sc->GetConstrainedVertexIndex2()) + current_vector;
		}
		if (cj->constraintType == ATTACHMENT) {
			// Work out attachment constraint.
		}
		//cj->m_RHS.applyThisOnTheLeft(*p_j);
		*b += *p_j;
	}*/

}

// h -> defines host
// d -> defines device
//Return b 
void Converge::Converge(double* h_spring, double* h_attach, double* h_pj, double* h_qn1, double* h_b) {

	double *d_b, *d_pj, *d_qn1, *d_pspring, *d_pattach; //Device memory.
	//Constraint* d_cj;
	//spring.data 
	//Instead of directly using Eigen use .data and conver it to a float3

	d_b = h_b;
	d_pj = h_pj;
	d_qn1 = h_qn1;
	d_pspring = h_spring;
	d_pattach = h_attach;


	//hipMalloc((void**)&d_cj, sizeof(Constraint));
	hipMalloc((void**)&d_b, sizeof(double*));
	hipMalloc((void**)&d_pj, sizeof(double*));
	//Pspring
	hipMalloc((void**)&d_pspring, sizeof(double*));
	//pAttach
	hipMalloc((void**)&d_pattach, sizeof(double*));
	//qn1
	hipMalloc((void**)&d_qn1, sizeof(double*));
	

	//hipMemcpy(d_cj, &cj, sizeof(Constraint), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(d_pj, h_pj, sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(d_pspring, h_spring, sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(d_pattach, h_attach, sizeof(double*), hipMemcpyHostToDevice);
	hipMemcpy(d_qn1, h_qn1, sizeof(double*), hipMemcpyHostToDevice);

	localStep<<<1, SIZE >>>(
		d_pspring,
		d_pattach, 
		d_pj, 
		d_qn1,
		d_b
		);
	
	hipDeviceSynchronize();
}
	

//Will need a function that will convert position and velocity to float3
//Returning of p_j will be needed. To be used on the global solver.
